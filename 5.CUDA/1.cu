#include "hip/hip_runtime.h"
/**
 * @file 2-1.cu
 * @author yan
 * @brief 向量加法
 *        cuda程序模板
 * @version 0.1
 * @date 2022-07-18
 * 
 * @copyright Copyright (c) 2022
 * 
 */


#include <iostream>

__global__ void add(float* A, float* B, float* C, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		C[i] = A[i] + B[i];
	}
}

int main()
{
	// 1.申请Host内存
	int N;
	N = 10000;
	float* A_h = NULL;
	float* B_h = NULL;
	float* C_h = NULL;
	hipHostMalloc((void**)&A_h, N * sizeof(float));
	hipHostMalloc((void**)&B_h, N * sizeof(float));
	hipHostMalloc((void**)&C_h, N * sizeof(float));

	// 2.申请Device内存
	float* A_d = NULL;
	float* B_d = NULL;
	float* C_d = NULL;
	hipMalloc((void**)&A_d, N * sizeof(float));
	hipMalloc((void**)&B_d, N * sizeof(float));
	hipMalloc((void**)&C_d, N * sizeof(float));

	// 3.初始化
	for (int i = 0; i != N; ++i)
	{
		A_h[i] = i;
		B_h[i] = i;
	}

	// 4.将Host数据拷贝到Device中
	hipMemcpy(A_d, A_h, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, N * sizeof(float), hipMemcpyHostToDevice);

	// 5.Kernel函数
	dim3 dg(ceil((float)N / 11), 1, 1);
	dim3 db(11, 1, 1);
	add << <dg, db >> > (A_d, B_d, C_d, N);

	// 6.将Device数据拷贝到Host中
	hipMemcpy(C_h, C_d, N * sizeof(float), hipMemcpyDeviceToHost);

	// 7.检测结果
	for (int i = 0; i != N; ++i)
	{
		if (C_h[i] != 2 * i)
		{
			std::cout << "helloworld" << std::endl;
		}
	}

	// 8.释放内存
	hipHostFree(A_h);
	hipHostFree(B_h);
	hipHostFree(C_h);
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

	return 0;
}