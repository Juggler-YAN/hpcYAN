#include "hip/hip_runtime.h"
/**
 * @file 2-2.cu
 * @author yan
 * @brief 矩阵相加
 *        高维数据转一维数据
 * @version 0.1
 * @date 2022-07-18
 * 
 * @copyright Copyright (c) 2022
 * 
 */


#include <iostream>

__global__ void add(float* A, float* B, float* C, int M, int N)
{ 
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < M && j < N)
	{
		C[i * N + j] = A[i * N + j] + B[i * N + j];
	}
}

int main()
{
	// 1.申请Host内存
	int M, N;
	M = 30;
	N = 37;
	float* A_h = NULL;
	float* B_h = NULL;
	float* C_h = NULL;
	hipHostMalloc((void**)&A_h, M * N * sizeof(float));
	hipHostMalloc((void**)&B_h, M * N * sizeof(float));
	hipHostMalloc((void**)&C_h, M * N * sizeof(float));

	// 2.申请Device内存
	float* A_d = NULL;
	float* B_d = NULL;
	float* C_d = NULL;
	hipMalloc((void**)&A_d, M * N * sizeof(float));
	hipMalloc((void**)&B_d, M * N * sizeof(float));
	hipMalloc((void**)&C_d, M * N * sizeof(float));

	// 3.初始化
	for (int i = 0; i != M; ++i)
	{
		for (int j = 0; j != N; ++j)
		{
			A_h[i * N + j] = i * N + j;
			B_h[i * N + j] = i * N + j;
		}
	}

	// 4.将Host数据拷贝到Device中
	hipMemcpy(A_d, A_h, M * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, M * N * sizeof(float), hipMemcpyHostToDevice);

	// 5.Kernel函数
	dim3 dg(ceil((float)N / 8.0), ceil((float)M / 8.0), 1);
	dim3 db(8, 8, 1);
	add << <dg, db >> > (A_d, B_d, C_d, M, N);

	// 6.将Device数据拷贝到Host中
	hipMemcpy(C_h, C_d, M * N * sizeof(float), hipMemcpyDeviceToHost);

	// 7.检测结果
	for (int i = 0; i != M; ++i)
	{
		for (int j = 0; j != N; ++j)
		{
			if (C_h[i * N + j] != 2 * (i * N + j))
			{
				std::cout << "helloworld" << std::endl;
			}
		}
	}

	// 8.释放内存
	hipHostFree(A_h);
	hipHostFree(B_h);
	hipHostFree(C_h);
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

	return 0;
}